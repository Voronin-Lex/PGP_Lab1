#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define MINVAL 1e-7

#define CSC(call) {                                                   \
	 hipError_t err = call;                                             \
	 if(err!=hipSuccess)                                              \
	 {                                                                  \
		 fprintf(stderr, "CUDA error in file '%s' in line %i: %s.\n",   \
            __FILE__, __LINE__, hipGetErrorString(err));				 \
	 }                                                                    \
 } while (0)

   
__global__ void Permute(double* Dev_Mtr, int* i, int* k, int* Dev_size)
{
	int index=blockDim.x*blockIdx.x+threadIdx.x;

	if(index<*Dev_size)
	{
		double tmp=Dev_Mtr[index*(*Dev_size)+(*i)];
		Dev_Mtr[index*(*Dev_size)+(*i)]=Dev_Mtr[index*(*Dev_size)+(*k)];
		Dev_Mtr[index*(*Dev_size)+(*k)]=tmp;	
	}

}


__global__ void MaxElement(double* Mtr, int Size, int i, int*strnum)
{
	double MaxValue=Mtr[i*Size+i];
	*strnum=i;

	for(int k=i; k<Size; k++)
	  {

		  if(fabs(Mtr[i*(Size)+k])>fabs(MaxValue))
			  {
				  *strnum=*strnum+1;    //��� ��� ����������� ������
				  *strnum=k;
				  MaxValue=Mtr[i*(Size)+k]; 
		      }
	  }

	if(fabs(MaxValue)<MINVAL)   //���� ������������ ������� ���� ���������� ��������, �� ���������� -1 -> ������������ ����� 0 � ������� �� �����
	{
	  *strnum=-1;
	}

}

__global__ void Gaus(double* Mtr, int Size, int i)
{
	int index=blockIdx.x*blockDim.x+threadIdx.x;
	if(index>i && index< Size)
	{
		double particial = -Mtr[i*Size+index]/Mtr[i*Size+i];

        for(int z=i; z<Size; z++)                             
			{
				Mtr[z*Size+index]=Mtr[z*Size+index]+Mtr[z*Size+i]*particial;
			}		

	}
}

int main()
{
	int Size;
	int hostDet=1;    
	int HSTcountPerm=0;     //������� ������������ �� �����
	scanf("%d", &Size);

	if (Size==0) return 0;
    
	double *Mtr = (double*)malloc(Size*Size*sizeof(double));

	for(int i=0; i<Size; i++)
	{
		for(int j=0; j<Size; j++)
			scanf("%lf", &Mtr[j*Size+i]);
	}

	int* dev_Size;   //������ �������, ������� �������� �� ������
    double* dev_Mtr; // ���� ������� ������� �������� �� ������
	CSC(hipMalloc((void**)&dev_Size, sizeof(int)));
	CSC(hipMalloc((void**)&dev_Mtr, Size*Size*sizeof(double)));
	

	CSC(hipMemcpy(dev_Size, &Size, sizeof(int), hipMemcpyHostToDevice));
	CSC(hipMemcpy(dev_Mtr, Mtr, Size*Size*sizeof(double), hipMemcpyHostToDevice));

	int * Curr_str=NULL;
	int* New_Main_str=NULL;

	CSC(hipMalloc((void**)&Curr_str, sizeof(int)));
	CSC(hipMalloc((void**)&New_Main_str, sizeof(int)));
	
	int HostNewMainString=0;
	for(int i=0; i<Size; i++)
	{
		
		hipMemcpy(New_Main_str, &i, sizeof(int), hipMemcpyHostToDevice);
		MaxElement<<<1,1>>>(dev_Mtr, Size, i, New_Main_str);
		hipMemcpy(&HostNewMainString, New_Main_str, sizeof(int), hipMemcpyDeviceToHost);
		  
		if (HostNewMainString==-1) { hostDet=0; break;} 
		
			if(HostNewMainString!=i)
		{
			CSC(hipMemcpy(Curr_str, &i, sizeof(int), hipMemcpyHostToDevice));
			//CSC(cudaMemcpy(dev_Mtr, Mtr, Size*Size*sizeof(double), cudaMemcpyHostToDevice));
			Permute<<<100,100>>>(dev_Mtr, Curr_str, New_Main_str, dev_Size);
		    //CSC(cudaMemcpy(Mtr, dev_Mtr, Size*Size*sizeof(double), cudaMemcpyDeviceToHost));
			HSTcountPerm++;
		
		}

		Gaus<<<100,100>>>(dev_Mtr, Size, i);
	}

	double *ResMtr = (double*)malloc(Size*Size*sizeof(double));

	CSC(hipMemcpy(ResMtr, dev_Mtr, Size*Size*sizeof(double), hipMemcpyDeviceToHost));

	if(hostDet==0)
	{
		double ans=0;
		printf("%e ", ans);
		//system("pause");
		return 0;
	}
      
	double Det;
	int CountNegativeElements=0;

	if(ResMtr[0]<0) CountNegativeElements++;
	Det=log(fabs(ResMtr[0]));

	for(int i=1; i<Size; i++)
	{
		if(ResMtr[i*Size+i]<0) CountNegativeElements++;
		Det+=log(fabs(ResMtr[i*Size+i]));
	}

	Det=pow(exp(1.0), Det)*pow(-1.0, CountNegativeElements);

	if((HSTcountPerm % 2)!=0) Det*=-1;
	   
	printf("%e", Det);
	//getchar();	

    return 0;
}